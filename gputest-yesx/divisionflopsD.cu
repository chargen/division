#include "hip/hip_runtime.h"
#include "timer.h"
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <climits>
#include "hip/hip_runtime.h"

#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>

#define BLOCK_NUM 1024
#define THREAD_NUM 512
#define ITERATION_PER_THREAD 64
#define DIV_NUM_PER_ITERATION 64

#define GFLOPS(time) ((float)BLOCK_NUM*THREAD_NUM*ITERATION_PER_THREAD*DIV_NUM_PER_ITERATION/(time)/1e6)

__device__ double m[BLOCK_NUM*THREAD_NUM];
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ double divD_fisq(double x, double number)
{
    // fisq
    union Type{
        long long i;
        double y;
    } d;
    d.y  = number;
    d.i  = 0x5fe6eb50c7aa19f9 - ( d.i >> 1 );
    d.y  = d.y * ( 1.5 - ( 0.5 * number * d.y * d.y ) );

    // newton
    d.y = d.y * (2 - number * d.y);
    d.y = d.y * (2 - number * d.y);

    return x*d.y;
}

#define divD_direct(a, b) ((a)/(b))
//#define divF_fdividef(a,b) __fdividef((a), (b)) 
#define divD_add(a,b) ((a)+(b))

#define division(funcName, divF) \
__global__ void funcName() { \
    int tid = blockIdx.x * blockDim.x + threadIdx.x; \
	double x = tid; \
    for (int i = 0; i < ITERATION_PER_THREAD; i++) { \
		x += 0.0001; \
        double p1 = i + 20001.0; \
        p1 = divF(divF(divF(divF(divF(divF(divF(divF(p1, x+0.01), x+0.02), x+0.03), x+0.04), x+0.05), x+0.06), x+0.07), x+0.08); \
        p1 = divF(divF(divF(divF(divF(divF(divF(divF(p1, x+0.09), x+0.10), x+0.11), x+0.12), x+0.13), x+0.14), x+0.15), x+0.16); \
        p1 = divF(divF(divF(divF(divF(divF(divF(divF(p1, x+0.17), x+0.18), x+0.19), x+0.20), x+0.21), x+0.22), x+0.23), x+0.24); \
        p1 = divF(divF(divF(divF(divF(divF(divF(divF(p1, x+0.25), x+0.26), x+0.27), x+0.28), x+0.29), x+0.30), x+0.31), x+0.32); \
        p1 = divF(divF(divF(divF(divF(divF(divF(divF(p1, x+0.33), x+0.34), x+0.35), x+0.36), x+0.37), x+0.38), x+0.39), x+0.40); \
        p1 = divF(divF(divF(divF(divF(divF(divF(divF(p1, x+0.41), x+0.42), x+0.43), x+0.44), x+0.45), x+0.46), x+0.47), x+0.48); \
        p1 = divF(divF(divF(divF(divF(divF(divF(divF(p1, x+0.49), x+0.50), x+0.51), x+0.52), x+0.53), x+0.54), x+0.55), x+0.56); \
        p1 = divF(divF(divF(divF(divF(divF(divF(divF(p1, x+0.57), x+0.58), x+0.59), x+0.60), x+0.61), x+0.62), x+0.63), x+0.64); \
        m[tid] += p1; \
    } \
}

division(division_fisq, divD_fisq)
division(division_direct, divD_direct)
division(division_add, divD_add)


int main(int argc, char* argv[])
{
    gpuErrchk(hipSetDevice(0));
    gpuErrchk(hipDeviceSetCacheConfig(hipFuncCachePreferL1));


    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );
    division_fisq<<<BLOCK_NUM, THREAD_NUM>>>();
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    printf("divD_fisq         time = %f ms, Gflops=%.2f \n", time, GFLOPS(time));

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );
    division_direct<<<BLOCK_NUM, THREAD_NUM>>>();
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    printf("divD_direct       time = %f ms, Gflops=%.2f \n", time, GFLOPS(time));

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );
    division_add<<<BLOCK_NUM, THREAD_NUM>>>();
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    printf("divD_add     time = %f ms, Gflops=%.2f \n", time, GFLOPS(time));

    hipEventDestroy( start );
    hipEventDestroy( stop );
    return 0;
}
